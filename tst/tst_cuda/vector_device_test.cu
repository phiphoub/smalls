#include "hip/hip_runtime.h"
#include "inc/vector.h"
#include "gtest/gtest.h"

using namespace smalls;

template<int SizeT, typename ScalarT>
__global__ void runAll(Vector<SizeT, ScalarT> vec1, Vector<SizeT, ScalarT> vec2)
{
    ScalarT scalar(3.0f);
    auto res1 = vec1 + vec2;
    res1 = vec2 - vec1;
    res1 = vec2 * vec1;
    res1 = vec2 / vec1;
    res1 = vec1 + scalar;
    res1 = vec2 - scalar;
    res1 = vec2 * scalar;
    res1 = vec2 / scalar;
    vec2 += vec1;
    vec2 -= vec1;
    vec2 *= vec1;
    vec2 /= vec1;
    vec1 += scalar;
    vec2 -= scalar;
    vec2 *= scalar;
    vec2 /= scalar;
    res1 = math::min(vec2, scalar);
    res1 = math::max(vec2,  scalar);
    res1 = math::min(vec2, vec1);
    res1 = math::max(vec2, vec1);
    res1 = math::floor(vec2);
    res1 = math::ceil(vec2);

    auto res2 = vec2 == vec1;
    res2 = vec2 != vec1;
    res2 = vec2 > vec1;
    res2 = vec2 < vec1;
    res2 = vec2 >= vec1;
    res2 = vec2 <= vec1;
    res2 = vec2 == scalar;
    res2 = vec2 != scalar;
    res2 = vec2 > scalar;
    res2 = vec2 < scalar;
    res2 = vec2 >= scalar;
    res2 = vec2 <= scalar;
}

TEST(VectorDevice, RunAllOperations)
{
    runAll << <1, 1 >> >(make_vector<float>(1, 2, 3, 4), make_vector<float>(5, 4, 3, 2));
    runAll << <1, 1 >> >(make_vector<double>(-2, 1), make_vector<double>(3, 2));
    runAll << <1, 1 >> >(
        make_vector<char>(1, 2, 3, 4, 5, 6, 7, 8, 9),
        make_vector<char>(9, 8, 7, 6, 5, 4, 3, 2, 1));
}


typedef Vector<4, float> VectorT;

__global__ void testGPU(const VectorT* dvec1, const VectorT* dvec2, VectorT* result)
{

    *result = (*dvec2) + (*dvec1);
}


TEST(VectorDevice, RunUploadDownloadOperation)
{
    VectorT vec1(1, 2, 3, 4);
    VectorT vec2(5, 4, 3, 2);
    VectorT result;

    VectorT *dvec1, *dvec2, *dres;
    hipMalloc((void**)&dvec1, sizeof(VectorT));
    hipMemcpy(dvec1, vec1.data, sizeof(VectorT), hipMemcpyHostToDevice);
    hipMalloc((void**)&dvec2, sizeof(VectorT));
    hipMemcpy(dvec2, vec2.data, sizeof(VectorT), hipMemcpyHostToDevice);

    hipMalloc((void**)&dres, sizeof(VectorT));

    testGPU<<<1, 1>>>(dvec1, dvec2, dres);
    hipMemcpy(result.data, dres->data, sizeof(VectorT), hipMemcpyDeviceToHost);

    EXPECT_EQ(6, result.data[0]);
    EXPECT_EQ(6, result.data[1]);
    EXPECT_EQ(6, result.data[2]);
    EXPECT_EQ(6, result.data[3]);
}
